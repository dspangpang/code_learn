
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square (float *d_out, float * d_in){
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f * f;
} 

int main(int argc, char** argv){
	const int ARRAY_SIZE = 8;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	
	//generate the input array on the host
	float h_in[ARRAY_SIZE];
	for(int i = 0; i < ARRAY_SIZE; i++){
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	//declear GPU memory pointer
	float * d_in;
	float * d_out;

	//allocate GPU memory
	hipMalloc((void**)&d_in, ARRAY_BYTES);
	hipMalloc((void**)&d_out, ARRAY_BYTES);

	//transfer the array to GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	//lanuch the kernel
	square<<<1,ARRAY_SIZE>>>(d_out, d_in);

	//copy the result to CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	//print the answer
	for(int j = 0; j < ARRAY_SIZE; j++){
		printf("%f\n",h_out[j]);
	}
	
	//free GPU memory
	hipFree(d_in);
	hipFree(d_out);
	
	return 0;

}
